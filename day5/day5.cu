#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdint.h>
#include "day5.cuh"

#include <string>
#include <vector>
#include <iostream>
#include <iomanip>
#include <sstream>
#include <ranges>
#include <algorithm>

#define MD5_BLOCK_SIZE 64
#define MD5_HASH_SIZE 16

// MD5 constants
__constant__ uint32_t k[64] = {
    0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee,
    0xf57c0faf, 0x4787c62a, 0xa8304613, 0xfd469501,
    0x698098d8, 0x8b44f7af, 0xffff5bb1, 0x895cd7be,
    0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821,
    0xf61e2562, 0xc040b340, 0x265e5a51, 0xe9b6c7aa,
    0xd62f105d, 0x02441453, 0xd8a1e681, 0xe7d3fbc8,
    0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed,
    0xa9e3e905, 0xfcefa3f8, 0x676f02d9, 0x8d2a4c8a,
    0xfffa3942, 0x8771f681, 0x6d9d6122, 0xfde5380c,
    0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70,
    0x289b7ec6, 0xeaa127fa, 0xd4ef3085, 0x04881d05,
    0xd9d4d039, 0xe6db99e5, 0x1fa27cf8, 0xc4ac5665,
    0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039,
    0x655b59c3, 0x8f0ccc92, 0xffeff47d, 0x85845dd1,
    0x6fa87e4f, 0xfe2ce6e0, 0xa3014314, 0x4e0811a1,
    0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391
};

__constant__ uint32_t r[64] = {
    7,12,17,22, 7,12,17,22, 7,12,17,22, 7,12,17,22,
    5,9,14,20, 5,9,14,20, 5,9,14,20, 5,9,14,20,
    4,11,16,23, 4,11,16,23, 4,11,16,23, 4,11,16,23,
    6,10,15,21, 6,10,15,21, 6,10,15,21, 6,10,15,21
};

__device__ __forceinline__ uint32_t leftrotate(uint32_t x, uint32_t c) {
    return (x << c) | (x >> (32 - c));
}

__device__ __forceinline__ uint32_t F(uint32_t x, uint32_t y, uint32_t z) { return (x & y) | (~x & z); }
__device__ __forceinline__ uint32_t G(uint32_t x, uint32_t y, uint32_t z) { return (x & z) | (y & ~z); }
__device__ __forceinline__ uint32_t H(uint32_t x, uint32_t y, uint32_t z) { return x ^ y ^ z; }
__device__ __forceinline__ uint32_t I(uint32_t x, uint32_t y, uint32_t z) { return y ^ (x | ~z); }

__device__ void lock(int *mutex) {
    while (atomicCAS(mutex, 0, 1) != 0) {
        // spin-wait
    }
}

__device__ void unlock(int *mutex) {
    atomicExch(mutex, 0);
}

__global__ void md5_kernel(
    const uint8_t* base_str,
    const uint32_t base_str_len,
    const uint32_t start_index,
    ANS* p1_a,
    ANS* p2_a,
    size_t num_inputs,
    int* mutex
) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_inputs) return;

    // Count digits of number to be appended
    uint32_t num = idx + start_index;
    uint32_t digits = 1;
    while (num >= 10) {
        num /= 10;
        digits++;
    }
    uint32_t len = base_str_len + digits;

    // Copy base string into new string
    uint8_t str[32] = {0};
    for (uint32_t i = 0; i < base_str_len && i < 32; i++) {
        str[i] = base_str[i];
    }

    // Add number on to end of input string
    int temp = idx + start_index;
    for (int i = len - 1; i >= (int)base_str_len; i--) {
        str[i] = '0' + (temp % 10);
        temp /= 10;
    }

    // Prepare single 64-byte block with padding
    uint8_t block[MD5_BLOCK_SIZE];

    // Copy input string
    #pragma unroll
    for (int i = 0; i < 32; i++) {
        block[i] = (i < len) ? str[i] : 0;
    }
    // Append 0x80 byte after message
    if (len < 32) block[len] = 0x80;

    // Zero out remaining bytes until 56
    #pragma unroll
    for (int i = len + 1; i < 56; i++) {
        block[i] = 0;
    }

    // Append length in bits (little-endian) at block[56..63]
    uint64_t bit_len = (uint64_t)len * 8;
    block[56] = (uint8_t)(bit_len & 0xff);
    block[57] = (uint8_t)((bit_len >> 8) & 0xff);
    block[58] = (uint8_t)((bit_len >> 16) & 0xff);
    block[59] = (uint8_t)((bit_len >> 24) & 0xff);
    block[60] = (uint8_t)((bit_len >> 32) & 0xff);
    block[61] = (uint8_t)((bit_len >> 40) & 0xff);
    block[62] = (uint8_t)((bit_len >> 48) & 0xff);
    block[63] = (uint8_t)((bit_len >> 56) & 0xff);

    // Prepare 16 uint32 words from block (little endian)
    uint32_t M[16];
    #pragma unroll
    for (int i = 0; i < 16; i++) {
        M[i] = (uint32_t)block[i*4] |
              ((uint32_t)block[i*4 + 1] << 8) |
              ((uint32_t)block[i*4 + 2] << 16) |
              ((uint32_t)block[i*4 + 3] << 24);
    }

    // Initialize MD5 state
    uint32_t a0 = 0x67452301;
    uint32_t b0 = 0xefcdab89;
    uint32_t c0 = 0x98badcfe;
    uint32_t d0 = 0x10325476;

    uint32_t A = a0, B = b0, C = c0, D = d0;

    // MD5 main loop
    #pragma unroll
    for (int i = 0; i < 64; i++) {
        uint32_t F_val, g;
        if (i < 16) {
            F_val = F(B, C, D);
            g = i;
        } else if (i < 32) {
            F_val = G(B, C, D);
            g = (5 * i + 1) % 16;
        } else if (i < 48) {
            F_val = H(B, C, D);
            g = (3 * i + 5) % 16;
        } else {
            F_val = I(B, C, D);
            g = (7 * i) % 16;
        }

        uint32_t temp = D;
        D = C;
        C = B;
        B = B + leftrotate((A + F_val + k[i] + M[g]), r[i]);
        A = temp;
    }

    a0 += A;
    b0 += B;
    c0 += C;
    d0 += D;

    // Store hash output (little endian)
    uint8_t out[MD5_HASH_SIZE];
    #pragma unroll
    for (int i = 0; i < 4; i++) {
        out[i]      = (a0 >> (8 * i)) & 0xff;
        out[i + 4]  = (b0 >> (8 * i)) & 0xff;
        out[i + 8]  = (c0 >> (8 * i)) & 0xff;
        out[i + 12] = (d0 >> (8 * i)) & 0xff;
    }

    if(!(*out) && !(*(out + 1)) && *(out + 2) < (uint8_t)16){
        
        lock(mutex);

        for(int j = 0; j < 8; j++){

            // Insert value into p1_a in the correct position based on indexes
            if((*p1_a).ans[j].index == -1){
                (*p1_a).ans[j].index = idx + start_index;
                (*p1_a).ans[j].value = *(out + 2);
                break;
            }else if((*p1_a).ans[j].index > (idx + start_index)){
                POS temp1 = POS{*(out + 2), static_cast<int32_t>(idx + start_index)};
                POS temp2 = (*p1_a).ans[j];
                do{
                    (*p1_a).ans[j] = temp1;
                    temp1 = temp2;
                    temp2 = (*p1_a).ans[j + 1 < 8 ? j + 1 : 7];
                    j++;
                } while (j < 8);
                break;
            }
 

        }

        // Insert into correct indexes in p2
        if(*(out + 2) < 8 && ((*p2_a).ans[*(out + 2)].index == -1 || (*p2_a).ans[*(out + 2)].index > (idx + start_index))){
            (*p2_a).ans[*(out + 2)].index = idx + start_index;
            (*p2_a).ans[*(out + 2)].value = *(out + 3) >> 4;
        }


        unlock(mutex);
    }

}

// Print md5 hash as a byte string
std::string md5_bytes_to_hex_string(const uint8_t* digest, size_t length = 16) {
    std::ostringstream oss;
    oss << std::hex << std::setfill('0');
    for (size_t i = 0; i < length; ++i) {
        oss << std::setw(2) << static_cast<int>(digest[i]);
    }
    return oss.str();
}

std::pair<std::string, std::string> md5(const str& input) {
    // Batch size
    constexpr int num_inputs = 5000000;
    int starting_index = 0;

    // Pointer device memory to hold input string
    uint8_t* d_input_str;

    int* d_mutex;

    // Output on host memory and pointers to output in device memory
    ANS p1_ans, p2_ans;
    ANS* d_p1_ans;
    ANS* d_p2_ans;

    // Initialise output values
    for(int i = 0; i < 8; i++){
        p1_ans.ans[i].index = -1;
        p2_ans.ans[i].index = -1;
        p1_ans.ans[i].value = 0;
        p2_ans.ans[i].value = 0;
    }

    // Allocate required memory
    hipMalloc(&d_mutex, sizeof(int));
    hipMalloc(&d_input_str, input.size);
    hipMalloc(&d_p1_ans, sizeof(ANS));
    hipMalloc(&d_p2_ans, sizeof(ANS));

    // Copy data to device
    hipMemcpy(d_input_str, input.data, input.size, hipMemcpyHostToDevice);
    hipMemcpy(d_p1_ans, &p1_ans, sizeof(ANS), hipMemcpyHostToDevice);
    hipMemcpy(d_p2_ans, &p2_ans, sizeof(ANS), hipMemcpyHostToDevice);

    bool done = false;

    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the maximum occupancy
    int gridSize;    // The actual grid size needed, based on input size

    // Calculate best block size and grid size
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, md5_kernel, 0, 0);

    if(blockSize == 0){
        blockSize = 256; // Fallback to a default block size
    }

    gridSize = (num_inputs + blockSize - 1) / blockSize;

    do{
        // Launch kernel
        md5_kernel<<<gridSize, blockSize>>>(
            d_input_str,
            input.size,
            starting_index,
            d_p1_ans,
            d_p2_ans,
            num_inputs,
            d_mutex
        );

        // Check for errors in kernel launch
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
            break;
        }

        // Wait for device to finish
        hipDeviceSynchronize();

        // Copy answers back to host memory
        hipMemcpy(&p1_ans, d_p1_ans, sizeof(ANS), hipMemcpyDeviceToHost);
        hipMemcpy(&p2_ans, d_p2_ans, sizeof(ANS), hipMemcpyDeviceToHost);

        starting_index += num_inputs;

        // Check if answers have been found
        done = true;
        for(int i = 0; i < 8; i++){
            if(p1_ans.ans[i].index == -1 || p2_ans.ans[i].index == -1){
                done = false;
                break;
            }
        }


    }while(!done);

    // Free device memory
    hipFree(d_input_str);
    hipFree(d_p1_ans);
    hipFree(d_p2_ans);
    hipFree(d_mutex);

    // Read out p1 answer
    std::string p1_str = "";
    for(int i = 0; i < 8; i++){
        p1_str += p1_ans.ans[i].value < 10 ? std::to_string(p1_ans.ans[i].value) : std::string(1, 'a' + (p1_ans.ans[i].value - 10));
    }

    // Read out p2 answer
    std::string p2_str = "";
    for(int i = 0; i < 8; i++){
        p2_str += p2_ans.ans[i].value < 10 ? std::to_string(p2_ans.ans[i].value) : std::string(1, 'a' + (p2_ans.ans[i].value - 10));
    }

    return {p1_str, p2_str};
}