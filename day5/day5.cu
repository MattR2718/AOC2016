#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdint.h>
#include "day5.cuh"

#include <string>
#include <vector>
#include <iostream>
#include <iomanip>
#include <sstream>
#include <ranges>
#include <algorithm>

#define MD5_BLOCK_SIZE 64
#define MD5_HASH_SIZE 16

// MD5 constants
__constant__ uint32_t k[64] = {
    0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee,
    0xf57c0faf, 0x4787c62a, 0xa8304613, 0xfd469501,
    0x698098d8, 0x8b44f7af, 0xffff5bb1, 0x895cd7be,
    0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821,
    0xf61e2562, 0xc040b340, 0x265e5a51, 0xe9b6c7aa,
    0xd62f105d, 0x02441453, 0xd8a1e681, 0xe7d3fbc8,
    0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed,
    0xa9e3e905, 0xfcefa3f8, 0x676f02d9, 0x8d2a4c8a,
    0xfffa3942, 0x8771f681, 0x6d9d6122, 0xfde5380c,
    0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70,
    0x289b7ec6, 0xeaa127fa, 0xd4ef3085, 0x04881d05,
    0xd9d4d039, 0xe6db99e5, 0x1fa27cf8, 0xc4ac5665,
    0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039,
    0x655b59c3, 0x8f0ccc92, 0xffeff47d, 0x85845dd1,
    0x6fa87e4f, 0xfe2ce6e0, 0xa3014314, 0x4e0811a1,
    0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391
};

__constant__ uint32_t r[64] = {
    7,12,17,22, 7,12,17,22, 7,12,17,22, 7,12,17,22,
    5,9,14,20, 5,9,14,20, 5,9,14,20, 5,9,14,20,
    4,11,16,23, 4,11,16,23, 4,11,16,23, 4,11,16,23,
    6,10,15,21, 6,10,15,21, 6,10,15,21, 6,10,15,21
};

__device__ __forceinline__ uint32_t leftrotate(uint32_t x, uint32_t c) {
    return (x << c) | (x >> (32 - c));
}

__device__ __forceinline__ uint32_t F(uint32_t x, uint32_t y, uint32_t z) { return (x & y) | (~x & z); }
__device__ __forceinline__ uint32_t G(uint32_t x, uint32_t y, uint32_t z) { return (x & z) | (y & ~z); }
__device__ __forceinline__ uint32_t H(uint32_t x, uint32_t y, uint32_t z) { return x ^ y ^ z; }
__device__ __forceinline__ uint32_t I(uint32_t x, uint32_t y, uint32_t z) { return y ^ (x | ~z); }

__device__ void lock(int *mutex) {
    while (atomicCAS(mutex, 0, 1) != 0) {
        // spin-wait
    }
}

__device__ void unlock(int *mutex) {
    atomicExch(mutex, 0);
}

__global__ void md5_kernel_variable_length(
    const uint8_t* base_str,
    const uint32_t base_str_len,
    const uint32_t start_index,
    uint8_t* __restrict__ output_hashes,
    size_t num_inputs,
    bool* valid_flags
) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_inputs) return;

    uint32_t num = idx + start_index;
    uint32_t digits = 1;
    while (num >= 10) {
        num /= 10;
        digits++;
    }
    uint32_t len = base_str_len + digits;

    uint8_t str[32] = {0};
    for (uint32_t i = 0; i < base_str_len && i < 32; i++) {
        str[i] = base_str[i];
    }

    int temp = idx + start_index;
    for (int i = len - 1; i >= (int)base_str_len; i--) {
        str[i] = '0' + (temp % 10);
        temp /= 10;
    }

    // Prepare single 64-byte block with padding
    uint8_t block[MD5_BLOCK_SIZE];

    // Copy input string
    #pragma unroll
    for (int i = 0; i < 32; i++) {
        block[i] = (i < len) ? str[i] : 0;
    }
    // Append 0x80 byte after message
    if (len < 32) block[len] = 0x80;

    // Zero out remaining bytes until 56
    #pragma unroll
    for (int i = len + 1; i < 56; i++) {
        block[i] = 0;
    }

    // Append length in bits (little-endian) at block[56..63]
    uint64_t bit_len = (uint64_t)len * 8;
    block[56] = (uint8_t)(bit_len & 0xff);
    block[57] = (uint8_t)((bit_len >> 8) & 0xff);
    block[58] = (uint8_t)((bit_len >> 16) & 0xff);
    block[59] = (uint8_t)((bit_len >> 24) & 0xff);
    block[60] = (uint8_t)((bit_len >> 32) & 0xff);
    block[61] = (uint8_t)((bit_len >> 40) & 0xff);
    block[62] = (uint8_t)((bit_len >> 48) & 0xff);
    block[63] = (uint8_t)((bit_len >> 56) & 0xff);

    // Prepare 16 uint32 words from block (little endian)
    uint32_t M[16];
    #pragma unroll
    for (int i = 0; i < 16; i++) {
        M[i] = (uint32_t)block[i*4] |
              ((uint32_t)block[i*4 + 1] << 8) |
              ((uint32_t)block[i*4 + 2] << 16) |
              ((uint32_t)block[i*4 + 3] << 24);
    }

    // Initialize MD5 state
    uint32_t a0 = 0x67452301;
    uint32_t b0 = 0xefcdab89;
    uint32_t c0 = 0x98badcfe;
    uint32_t d0 = 0x10325476;

    uint32_t A = a0, B = b0, C = c0, D = d0;

    // MD5 main loop
    #pragma unroll
    for (int i = 0; i < 64; i++) {
        uint32_t F_val, g;
        if (i < 16) {
            F_val = F(B, C, D);
            g = i;
        } else if (i < 32) {
            F_val = G(B, C, D);
            g = (5 * i + 1) % 16;
        } else if (i < 48) {
            F_val = H(B, C, D);
            g = (3 * i + 5) % 16;
        } else {
            F_val = I(B, C, D);
            g = (7 * i) % 16;
        }

        uint32_t temp = D;
        D = C;
        C = B;
        B = B + leftrotate((A + F_val + k[i] + M[g]), r[i]);
        A = temp;
    }

    a0 += A;
    b0 += B;
    c0 += C;
    d0 += D;

    // Store hash output (little endian)
    uint8_t* out = output_hashes + idx * MD5_HASH_SIZE;
    #pragma unroll
    for (int i = 0; i < 4; i++) {
        out[i]      = (a0 >> (8 * i)) & 0xff;
        out[i + 4]  = (b0 >> (8 * i)) & 0xff;
        out[i + 8]  = (c0 >> (8 * i)) & 0xff;
        out[i + 12] = (d0 >> (8 * i)) & 0xff;
    }

    if(!(*out) && !(*(out + 1)) && *(out + 2) < (uint8_t)16){
        *(valid_flags + idx) = true;
    }

}

std::string md5_bytes_to_hex_string(const uint8_t* digest, size_t length = 16) {
    std::ostringstream oss;
    oss << std::hex << std::setfill('0');
    for (size_t i = 0; i < length; ++i) {
        oss << std::setw(2) << static_cast<int>(digest[i]);
    }
    return oss.str();
}

void md5(const str& input, str& p1, str& p2) {
    constexpr int num_inputs = 30000000;
    int starting_index = 0;

    std::cout<<"MAPPING TO DEVICE MEMORY\n";

    // Map data into device memory
    uint8_t* d_input_str;
    uint8_t* d_output_hashes;
    bool* d_valid_flags;

    ANS p1_ans, p2_ans;

    
    hipMalloc(&d_input_str, input.size);
    hipMalloc(&d_output_hashes, num_inputs * MD5_HASH_SIZE);
    hipMalloc(&d_valid_flags, num_inputs * sizeof(bool));

    hipMemcpy(d_input_str, input.data, input.size, hipMemcpyHostToDevice);
    hipMemset(d_valid_flags, 0, num_inputs * sizeof(bool));

    std::cout<<"LAUNCHING KERNEL\n";

    // Launch kernel
    int blockSize = 256;
    int numBlocks = (num_inputs + blockSize - 1) / blockSize;
    md5_kernel_variable_length<<<numBlocks, blockSize>>>(
        d_input_str,
        input.size,
        starting_index,
        d_output_hashes,
        num_inputs,
        d_valid_flags
    );

    hipDeviceSynchronize();

    std::cout<<"COPYING OUTPUT HASHES BACK TO HOST\n";

    // Copy output hashes back to host
    std::vector<uint8_t> output_hashes(num_inputs * MD5_HASH_SIZE);
    hipMemcpy(output_hashes.data(), d_output_hashes, output_hashes.size(), hipMemcpyDeviceToHost);

    std::vector<uint8_t> valid_flags(num_inputs);
    hipMemcpy(valid_flags.data(), d_valid_flags, valid_flags.size(), hipMemcpyDeviceToHost);

    std::cout<<"FREEING DEVICE MEMORY\n";

    // Free device memory
    hipFree(d_output_hashes);
    hipFree(d_input_str);
    hipFree(d_valid_flags);

    std::cout<<"DEVICE MEMORY FREED\n";

    // Process output_hashes to produce final results
    std::string p1_str = md5_bytes_to_hex_string((uint8_t*)output_hashes.data());            // first hash
    std::string p2_str = md5_bytes_to_hex_string((uint8_t*)output_hashes.data() + 16);       // second hash


    //for(int i = 0; i < output_hashes.size(); i += 16){
        //if(!(*(output_hashes.data() + i)) && !(*(output_hashes.data() + i + 1)) && *(output_hashes.data() + i + 2) < (uint8_t)16){
        //    std::cout << "\tFound matching hash: " << md5_bytes_to_hex_string((uint8_t*)output_hashes.data() + i) << '\n';
        //}
    //}
     for(int i = 0; i < valid_flags.size(); i++){

         if(valid_flags[i]){
             std::cout << "\tFound matching hash: " << md5_bytes_to_hex_string((uint8_t*)output_hashes.data() + i * MD5_HASH_SIZE) << '\n';
         }

     }


    p1.init(p1_str.c_str(), p1_str.size());
    p2.init(p2_str.c_str(), p2_str.size());

}